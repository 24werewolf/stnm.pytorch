#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include "stnm_cuda_kernel.h"

#define real float

// Bilinear sampling is done in BHWD (coalescing is not obvious in BDHW)
// we assume BHWD format in inputImages
// we assume BHW(YX) format on grids

__device__ void getTopLeft(float x, int width, int& point, float& weight)
{
   /* for interpolation :
      stores in point and weight :
      - the x-coordinate of the pixel on the left (or y-coordinate of the upper pixel)
      - the weight for interpolating
   */

   float xcoord = (x + 1) * (width - 1) / 2;
   point = floor(xcoord);
   weight = 1 - (xcoord - point);
}

__device__ bool between(int value, int lowerBound, int upperBound)
{
   return (value >= lowerBound && value <= upperBound);
}

__device__ void sumReduceShMem(volatile float s[])
{
   /* obviously only works for 32 elements */
   /* sums up a shared memory array of 32 elements, stores it in s[0] */
   /* whole warp can then read first element (broadcasting) */
   if(threadIdx.x<16) { s[threadIdx.x] = s[threadIdx.x] + s[threadIdx.x+16]; }
   if(threadIdx.x<8) { s[threadIdx.x] = s[threadIdx.x] + s[threadIdx.x+8]; }
   if(threadIdx.x<4) { s[threadIdx.x] = s[threadIdx.x] + s[threadIdx.x+4]; }
   if(threadIdx.x<2) { s[threadIdx.x] = s[threadIdx.x] + s[threadIdx.x+2]; }
   if(threadIdx.x<1) { s[threadIdx.x] = s[threadIdx.x] + s[threadIdx.x+1]; }
}

// __global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, int inputImages_strideHeight, int inputImages_strideWidth,
//                                          float* grids_data, int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth,
//                                          float* output_data, int output_strideBatch, int output_strideChannels, int output_strideHeight, int output_strideWidth,
//                                          int inputImages_channels, int inputImages_height, int inputImages_width, int output_width)
// {
//    // each (32,16) block 16 output pixels (for coalescing the grid read)
//    // x,y = coordinates (xOut = blockIdx.x*16+blockDim.y+threadIdx.y)
//    // z = batch index
//    // threadIdx.x : used for features (coalescing is trivial)
//
//    const int xOut = blockIdx.x*blockDim.y+threadIdx.y;
//    const bool withinImageBounds = xOut < output_width;
//    const bool withinGridBounds = blockIdx.x*blockDim.y + threadIdx.x / 2 < output_width;
//    const int yOut = blockIdx.y;
//    const int width = inputImages_width;
//    const int height = inputImages_height;
//
//    const int b = blockIdx.z;
//
//    float yf,xf;
//
//    __shared__ float gridData[32];
//    if (threadIdx.y==0 && withinGridBounds)
//    {
//       gridData[threadIdx.x] = grids_data[b*grids_strideBatch + yOut*grids_strideHeight + xOut*grids_strideWidth + threadIdx.x];
//    }
//    __syncthreads();
//    if(!withinImageBounds) return;
//    yf = gridData[threadIdx.y*2];
//    xf = gridData[threadIdx.y*2+1];
//
//    int yInTopLeft, xInTopLeft;
//    float yWeightTopLeft, xWeightTopLeft;
//    getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
//    getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);
//
//    const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;
//    const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
//    const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
//    const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
//    const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;
//
//    float v=0;
//    float inTopLeft=0;
//    float inTopRight=0;
//    float inBottomLeft=0;
//    float inBottomRight=0;
//
//    bool topLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft, 0, height-1);
//    bool topRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft, 0, height-1);
//    bool bottomLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft+1, 0, height-1);
//    bool bottomRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft+1, 0, height-1);
//
//    // interpolation happens here
//    for(int t=threadIdx.x; t<inputImages_channels; t+= blockDim.x)
//    {
//       if(topLeftIsIn) inTopLeft = inputImages_data[inTopLeftAddress + t];
//       if(topRightIsIn) inTopRight = inputImages_data[inTopRightAddress + t];
//       if(bottomLeftIsIn) inBottomLeft = inputImages_data[inBottomLeftAddress + t];
//       if(bottomRightIsIn) inBottomRight = inputImages_data[inBottomRightAddress + t];
//
//       v = xWeightTopLeft * yWeightTopLeft * inTopLeft
//         + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
//         + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
//         + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;
//
//       output_data[outAddress + t] = v;
//    }
// }

__global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, int inputImages_strideHeight, int inputImages_strideWidth,
                                         float* grids_data, int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth,
                                         float* masks_data, int masks_strideBatch, int masks_strideYX, int masks_strideHeight, int masks_strideWidth,
                                         float* canvas_data, int canvas_strideBatch, int canvas_strideYX, int canvas_strideHeight, int canvas_strideWidth,
                                         float* output_data, int output_strideBatch, int output_strideChannels, int output_strideHeight, int output_strideWidth,
                                         int inputImages_channels, int inputImages_height, int inputImages_width, int output_width)
{
   // each (32,16) block 16 output pixels (for coalescing the grid read)
   // x,y = coordinates (xOut = blockIdx.x*16+blockDim.y+threadIdx.y)
   // z = batch index
   // threadIdx.x : used for features (coalescing is trivial)

   const int xOut = blockIdx.x*blockDim.y+threadIdx.y;
   const bool withinImageBounds = xOut < output_width;
   const bool withinGridBounds = blockIdx.x*blockDim.y + threadIdx.x / 2 < output_width;
   const int yOut = blockIdx.y;
   const int width = inputImages_width;
   const int height = inputImages_height;

   const int b = blockIdx.z;

   float yf,xf;

   __shared__ float gridData[32];
   if (threadIdx.y==0 && withinGridBounds)
   {
     // #if __CUDA_ARCH__>=200
     //    printf("%d \n", grids_strideWidth);
     // #endif
      gridData[threadIdx.x] = grids_data[b*grids_strideBatch + yOut*grids_strideHeight + xOut*grids_strideWidth + threadIdx.x];
   }
   __syncthreads();
   if(!withinImageBounds) return;
   yf = gridData[threadIdx.y*2];
   xf = gridData[threadIdx.y*2+1];

   int yInTopLeft, xInTopLeft;
   float yWeightTopLeft, xWeightTopLeft;
   // if (xf < 1 && xf > -1 && yf < 1 && yf > -1)
   // printf("xf: %f, yf: %f\n", xf, yf);

   getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
   getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

   // xWeightTopLeft = 0.5;
   // yWeightTopLeft = 0.5;

   const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;

   const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
   const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
   const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
   const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

   const int inTopLeftMaskAddress = masks_strideBatch * b + masks_strideHeight * yInTopLeft + masks_strideWidth * xInTopLeft;
   const int inTopRightMaskAddress = inTopLeftMaskAddress + masks_strideWidth;
   const int inBottomLeftMaskAddress = inTopLeftMaskAddress + masks_strideHeight;
   const int inBottomRightMaskAddress = inBottomLeftMaskAddress + masks_strideWidth;

   float v=0;
   float inTopLeft=0;
   float inTopRight=0;
   float inBottomLeft=0;
   float inBottomRight=0;

   float m = 0;
   float inTopLeftMask=0;
   float inTopRightMask=0;
   float inBottomLeftMask=0;
   float inBottomRightMask=0;

   bool topLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft, 0, height-1);
   bool topRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft, 0, height-1);
   bool bottomLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft+1, 0, height-1);
   bool bottomRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft+1, 0, height-1);

   if(topLeftIsIn) inTopLeftMask = masks_data[inTopLeftMaskAddress];
   if(topRightIsIn) inTopRightMask = masks_data[inTopRightMaskAddress];
   if(bottomLeftIsIn) inBottomLeftMask = masks_data[inBottomLeftMaskAddress];
   if(bottomRightIsIn) inBottomRightMask = masks_data[inBottomRightMaskAddress];

   m = xWeightTopLeft * yWeightTopLeft * inTopLeftMask
     + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRightMask
     + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeftMask
     + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRightMask;

   // interpolation happens here
   for(int t=threadIdx.x; t<inputImages_channels; t+= blockDim.x)
   {
      // jw2yang: do not change output_data when it locates outside the source image,
      // Todo: check backward after considering this case.
      if (!topLeftIsIn && !topRightIsIn && !bottomLeftIsIn && !bottomRightIsIn)
        output_data[outAddress + t] = canvas_data[outAddress + t];

      if(topLeftIsIn) inTopLeft = inputImages_data[inTopLeftAddress + t];
      if(topRightIsIn) inTopRight = inputImages_data[inTopRightAddress + t];
      if(bottomLeftIsIn) inBottomLeft = inputImages_data[inBottomLeftAddress + t];
      if(bottomRightIsIn) inBottomRight = inputImages_data[inBottomRightAddress + t];

      v = xWeightTopLeft * yWeightTopLeft * inTopLeft
        + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
        + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
        + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

      // we do not replace the canvas region with foreground, instead, we add value together.
      // printf("mask value: %f\n", m);
      // printf("bg value: %f\n", canvas_data[outAddress + t]);
      // printf("fg value: %f\n", v);
      output_data[outAddress + t] = (1 - m) * canvas_data[outAddress + t] + m * v;
      // printf("out value: %f\n", output_data[outAddress + t]);
      // output_data[outAddress + t] = v;
   }
}

template<bool onlyGrid> __global__ void backwardBilinearSampling(
                                         float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, int inputImages_strideHeight, int inputImages_strideWidth,
                                         float* gradInputImages_data, int gradInputImages_strideBatch, int gradInputImages_strideChannels, int gradInputImages_strideHeight, int gradInputImages_strideWidth,
                                         float* grids_data, int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth,
                                         float* gradGrids_data, int gradGrids_strideBatch, int gradGrids_strideYX, int gradGrids_strideHeight, int gradGrids_strideWidth,
                                         float* masks_data, int masks_strideBatch, int masks_strideYX, int masks_strideHeight, int masks_strideWidth,
                                         float* gradMasks_data, int gradMasks_strideBatch, int gradMasks_strideYX, int gradMasks_strideHeight, int gradMasks_strideWidth,
                                         float* canvas_data, int canvas_strideBatch, int canvas_strideYX, int canvas_strideHeight, int canvas_strideWidth,
                                         float* gradCanvas_data, int gradCanvas_strideBatch, int gradCanvas_strideYX, int gradCanvas_strideHeight, int gradCanvas_strideWidth,
                                         float* gradOutput_data, int gradOutput_strideBatch, int gradOutput_strideChannels, int gradOutput_strideHeight, int gradOutput_strideWidth,
                                         int inputImages_channels, int inputImages_height, int inputImages_width, int gradOutput_width)
{
   // each (32,16) block 16 output pixels (for coalescing the grid read)
   // x,y = coordinates
   // z = batch index
   // threads : used for features

   const int xOut = blockIdx.x*blockDim.y+threadIdx.y;
   const bool withinImageBounds = xOut < gradOutput_width;
   const bool withinGridBounds = blockIdx.x*blockDim.y + threadIdx.x / 2 < gradOutput_width;

   const int yOut = blockIdx.y;
   const int width = inputImages_width;
   const int height = inputImages_height;

   const int b = blockIdx.z;

   float yf,xf;

   __shared__ float gridData[32];
   if (threadIdx.y==0 && withinGridBounds)
   {
      gridData[threadIdx.x] = grids_data[b*grids_strideBatch + yOut*grids_strideHeight + xOut*grids_strideWidth + threadIdx.x];
   }
   __syncthreads();

   if(withinImageBounds)
   {
      yf = gridData[threadIdx.y*2];
      xf = gridData[threadIdx.y*2+1];

      int yInTopLeft, xInTopLeft;
      float yWeightTopLeft, xWeightTopLeft;
      getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
      getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

      // xWeightTopLeft = 0.5;
      // yWeightTopLeft = 0.5;

      const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
      const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
      const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
      const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

      const int gradInputImagesTopLeftAddress = gradInputImages_strideBatch * b + gradInputImages_strideHeight * yInTopLeft + gradInputImages_strideWidth * xInTopLeft;
      const int gradInputImagesTopRightAddress = gradInputImagesTopLeftAddress + gradInputImages_strideWidth;
      const int gradInputImagesBottomLeftAddress = gradInputImagesTopLeftAddress + gradInputImages_strideHeight;
      const int gradInputImagesBottomRightAddress = gradInputImagesBottomLeftAddress + gradInputImages_strideWidth;

      const int inTopLeftMaskAddress = masks_strideBatch * b + masks_strideHeight * yInTopLeft + masks_strideWidth * xInTopLeft;
      const int inTopRightMaskAddress = inTopLeftMaskAddress + masks_strideWidth;
      const int inBottomLeftMaskAddress = inTopLeftMaskAddress + masks_strideHeight;
      const int inBottomRightMaskAddress = inBottomLeftMaskAddress + masks_strideWidth;

      const int gradMasksTopLeftAddress = gradMasks_strideBatch * b + gradMasks_strideHeight * yInTopLeft + gradMasks_strideWidth * xInTopLeft;
      const int gradMasksTopRightAddress = gradMasksTopLeftAddress + gradMasks_strideWidth;
      const int gradMasksBottomLeftAddress = gradMasksTopLeftAddress + gradMasks_strideHeight;
      const int gradMasksBottomRightAddress = gradMasksBottomLeftAddress + gradMasks_strideWidth;

      const int gradOutputAddress = gradOutput_strideBatch * b + gradOutput_strideHeight * yOut + gradOutput_strideWidth * xOut;

      float topLeftDotProduct = 0;
      float topRightDotProduct = 0;
      float bottomLeftDotProduct = 0;
      float bottomRightDotProduct = 0;

      bool topLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft, 0, height-1);
      bool topRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft, 0, height-1);
      bool bottomLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft+1, 0, height-1);
      bool bottomRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft+1, 0, height-1);

      float v = 0;
      float inTopLeft=0;
      float inTopRight=0;
      float inBottomLeft=0;
      float inBottomRight=0;

      float c = 0;

      float m = 0;
      float inTopLeftMask=0;
      float inTopRightMask=0;
      float inBottomLeftMask=0;
      float inBottomRightMask=0;

      if(topLeftIsIn) inTopLeftMask = masks_data[inTopLeftMaskAddress];
      if(topRightIsIn) inTopRightMask = masks_data[inTopRightMaskAddress];
      if(bottomLeftIsIn) inBottomLeftMask = masks_data[inBottomLeftMaskAddress];
      if(bottomRightIsIn) inBottomRightMask = masks_data[inBottomRightMaskAddress];

      m = xWeightTopLeft * yWeightTopLeft * inTopLeftMask
        + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRightMask
        + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeftMask
        + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRightMask;

      /*
         In that loop we accumulate
         - gradients into the gradInputImages array with atomic adds
         - we compute the dot product that we need for the grid gradient
      */

      for(int t=threadIdx.x; t<inputImages_channels; t+= blockDim.x)
      {
         float gradOutValue = gradOutput_data[gradOutputAddress + t];
         float gradOutValue_fg = gradOutValue * m;
         float gradOutValue_bg = gradOutValue * (1 - m);

         // jw2yang: copy the gradients outside the object region to canvas, and inside region
         if (!topLeftIsIn && !topRightIsIn && !bottomLeftIsIn && !bottomRightIsIn) {
            gradCanvas_data[gradOutputAddress + t] = gradOutput_data[gradOutputAddress + t];
            // if (b == 0 && yOut == 10 && xOut == 10) {
            //    printf("all out");
            //    printf("gradOut value: %f ", gradOutput_data[gradOutputAddress + t]);
            //    printf("gradCanvas value: %f ", gradCanvas_data[gradOutputAddress + t]);
            // }
         }
         else {
           gradCanvas_data[gradOutputAddress + t] = gradOutValue_bg;
          //  if (b == 0 && yOut == 10 && xOut == 10) {
          //     printf("gradOut value: %f ", gradOutput_data[gradOutputAddress + t]);
          //     printf("gradCanvas value: %f ", gradCanvas_data[gradOutputAddress + t]);
          //  }
         }

         // bool between(int value, int lowerBound, int upperBound)
         if(topLeftIsIn)
         {
            float inTopLeft = inputImages_data[inTopLeftAddress + t];
            topLeftDotProduct += inTopLeft * gradOutValue_fg;
            if(!onlyGrid) atomicAdd(&gradInputImages_data[gradInputImagesTopLeftAddress + t], xWeightTopLeft * yWeightTopLeft * gradOutValue_fg);
         }

         if(topRightIsIn)
         {
            float inTopRight = inputImages_data[inTopRightAddress + t];
            topRightDotProduct += inTopRight * gradOutValue_fg;
            if(!onlyGrid) atomicAdd(&gradInputImages_data[gradInputImagesTopRightAddress + t], (1 - xWeightTopLeft) * yWeightTopLeft * gradOutValue_fg);
         }

         if(bottomLeftIsIn)
         {
            float inBottomLeft = inputImages_data[inBottomLeftAddress + t];
            bottomLeftDotProduct += inBottomLeft * gradOutValue_fg;
            if(!onlyGrid) atomicAdd(&gradInputImages_data[gradInputImagesBottomLeftAddress + t], xWeightTopLeft * (1 - yWeightTopLeft) * gradOutValue_fg);
         }

         if(bottomRightIsIn)
         {
            float inBottomRight = inputImages_data[inBottomRightAddress + t];
            bottomRightDotProduct += inBottomRight * gradOutValue_fg;
            if(!onlyGrid) atomicAdd(&gradInputImages_data[gradInputImagesBottomRightAddress + t], (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * gradOutValue_fg);
         }

         // jw2yang: compute the gradient mask value
         if(topLeftIsIn) inTopLeft = inputImages_data[inTopLeftAddress + t];
         if(topRightIsIn) inTopRight = inputImages_data[inTopRightAddress + t];
         if(bottomLeftIsIn) inBottomLeft = inputImages_data[inBottomLeftAddress + t];
         if(bottomRightIsIn) inBottomRight = inputImages_data[inBottomRightAddress + t];
         v = xWeightTopLeft * yWeightTopLeft * inTopLeft
           + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
           + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
           + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

         c = canvas_data[gradOutputAddress + t];

         float gradMaskValue = gradOutValue * (v - c);


         // update gradient on mask map
         if(topLeftIsIn)
         {
            if(!onlyGrid) atomicAdd(&gradMasks_data[gradMasksTopLeftAddress], xWeightTopLeft * yWeightTopLeft * gradMaskValue);
         }

         if(topRightIsIn)
         {
            if(!onlyGrid) atomicAdd(&gradMasks_data[gradMasksTopRightAddress], (1 - xWeightTopLeft) * yWeightTopLeft * gradMaskValue);
         }

         if(bottomLeftIsIn)
         {
            if(!onlyGrid) atomicAdd(&gradMasks_data[gradMasksBottomLeftAddress], xWeightTopLeft * (1 - yWeightTopLeft) * gradMaskValue);
         }

         if(bottomRightIsIn)
         {
            if(!onlyGrid) atomicAdd(&gradMasks_data[gradMasksBottomRightAddress], (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * gradMaskValue);
         }
      }
      /*
         Here we reduce the dot product and compute the grid gradient before writing it.
      */

      /* could do shuffles and use no shmem at all but cuda arch is 2.0 */
      __shared__ volatile float __shmem[16][32];
      __shmem[threadIdx.y][threadIdx.x] = topLeftDotProduct;
      sumReduceShMem(__shmem[threadIdx.y]);
      topLeftDotProduct = __shmem[threadIdx.y][0];

      __shmem[threadIdx.y][threadIdx.x] = topRightDotProduct;
      sumReduceShMem(__shmem[threadIdx.y]);
      topRightDotProduct = __shmem[threadIdx.y][0];

      __shmem[threadIdx.y][threadIdx.x] = bottomLeftDotProduct;
      sumReduceShMem(__shmem[threadIdx.y]);
      bottomLeftDotProduct = __shmem[threadIdx.y][0];

      __shmem[threadIdx.y][threadIdx.x] = bottomRightDotProduct;
      sumReduceShMem(__shmem[threadIdx.y]);
      bottomRightDotProduct = __shmem[threadIdx.y][0];

      yf = - xWeightTopLeft * topLeftDotProduct + xWeightTopLeft * bottomLeftDotProduct - (1-xWeightTopLeft) * topRightDotProduct + (1-xWeightTopLeft) * bottomRightDotProduct;
      xf = - yWeightTopLeft * topLeftDotProduct + yWeightTopLeft * topRightDotProduct - (1-yWeightTopLeft) * bottomLeftDotProduct + (1-yWeightTopLeft) * bottomRightDotProduct;

      if(threadIdx.x==0)
      {
         gridData[threadIdx.y*2] = yf * (inputImages_height-1) / 2;
         gridData[threadIdx.y*2+1] = xf * (inputImages_width-1) / 2;
      }
   }// must put a big if condition in order not to hang at __syncthreads()...
   __syncthreads();

   if(threadIdx.y==0 && withinGridBounds)
       gradGrids_data[b*gradGrids_strideBatch + yOut*gradGrids_strideHeight + xOut*gradGrids_strideWidth + threadIdx.x] = gridData[threadIdx.x];
}

#ifdef __cplusplus
extern "C" {
#endif

int BilinearSamplerBHWD_updateOutput_cuda_kernel(/*output->size[2]*/int sz1,
                                                 /*output->size[1]*/int sz2,
                                                 /*output->size[0]*/int sz3,
                                                 float* inputImages_data,
                                                 int inputImages_strideBatch,
                                                 int inputImages_strideChannels,
                                                 int inputImages_strideHeight,
                                                 int inputImages_strideWidth,
                                                 float* grids_data,
                                                 int grids_strideBatch,
                                                 int grids_strideYX,
                                                 int grids_strideHeight,
                                                 int grids_strideWidth,
                                                 float* masks_data,
                                                 int masks_strideBatch,
                                                 int masks_strideYX,
                                                 int masks_strideHeight,
                                                 int masks_strideWidth,
                                                 float* canvas_data,
                                                 int canvas_strideBatch,
                                                 int canvas_strideYX,
                                                 int canvas_strideHeight,
                                                 int canvas_strideWidth,
                                                 float* output_data,
                                                 int output_strideBatch,
                                                 int output_strideChannels,
                                                 int output_strideHeight,
                                                 int output_strideWidth,
                                                 int inputImages_channels,
                                                 int inputImages_height,
                                                 int inputImages_width,
                                                 int output_width,
                                                 /*THCState_getCurrentStream(state)*/hipStream_t stream)
{
   //dim3 blocks((output->size[2]+15)/16, output->size[1], output->size[0]);
   dim3 blocks((sz1+15)/16, sz2, sz3);
   dim3 threads(32,16);

   /* assume BHWD */
   bilinearSamplingFromGrid <<< blocks, threads, 0, /*THCState_getCurrentStream(state)*/stream >>> (
     inputImages_data,
     inputImages_strideBatch,
     inputImages_strideChannels,
     inputImages_strideHeight,
     inputImages_strideWidth,
     grids_data,
     grids_strideBatch,
     grids_strideYX,
     grids_strideHeight,
     grids_strideWidth,
     masks_data,
     masks_strideBatch,
     masks_strideYX,
     masks_strideHeight,
     masks_strideWidth,
     canvas_data,
     canvas_strideBatch,
     canvas_strideYX,
     canvas_strideHeight,
     canvas_strideWidth,
     output_data,
     output_strideBatch,
     output_strideChannels,
     output_strideHeight,
     output_strideWidth,
     inputImages_channels,
     inputImages_height,
     inputImages_width,
     output_width);


  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in BilinearSampler.updateOutput: %s\n", hipGetErrorString(err));
    //THError("aborting");
    return 0;
  }
  return 1;
}

int BilinearSamplerBHWD_updateGradInput_cuda_kernel(/*gradOutput->size[2]*/int sz1,
                                                    /*gradOutput->size[1]*/int sz2,
                                                    /*gradOutput->size[0]*/int sz3,
                                                    float* inputImages_data,
                                                    int inputImages_strideBatch,
                                                    int inputImages_strideChannels,
                                                    int inputImages_strideHeight,
                                                    int inputImages_strideWidth,
                                                    float* gradInputImages_data,
                                                    int gradInputImages_strideBatch,
                                                    int gradInputImages_strideChannels,
                                                    int gradInputImages_strideHeight,
                                                    int gradInputImages_strideWidth,
                                                    float* grids_data,
                                                    int grids_strideBatch,
                                                    int grids_strideYX,
                                                    int grids_strideHeight,
                                                    int grids_strideWidth,
                                                    float* gradGrids_data,
                                                    int gradGrids_strideBatch,
                                                    int gradGrids_strideYX,
                                                    int gradGrids_strideHeight,
                                                    int gradGrids_strideWidth,
                                                    float* masks_data,
                                                    int masks_strideBatch,
                                                    int masks_strideYX,
                                                    int masks_strideHeight,
                                                    int masks_strideWidth,
                                                    float* gradMasks_data,
                                                    int gradMasks_strideBatch,
                                                    int gradMasks_strideYX,
                                                    int gradMasks_strideHeight,
                                                    int gradMasks_strideWidth,
                                                    float* canvas_data,
                                                    int canvas_strideBatch,
                                                    int canvas_strideYX,
                                                    int canvas_strideHeight,
                                                    int canvas_strideWidth,
                                                    float* gradCanvas_data,
                                                    int gradCanvas_strideBatch,
                                                    int gradCanvas_strideYX,
                                                    int gradCanvas_strideHeight,
                                                    int gradCanvas_strideWidth,
                                                    float* gradOutput_data,
                                                    int gradOutput_strideBatch,
                                                    int gradOutput_strideChannels,
                                                    int gradOutput_strideHeight,
                                                    int gradOutput_strideWidth,
                                                    int inputImages_channels,
                                                    int inputImages_height,
                                                    int inputImages_width,
                                                    int gradOutput_width,
                                                    /*THCState_getCurrentStream(state)*/hipStream_t stream)
{
//  THCState *state = getCutorchState(L);
//  THCudaTensor *inputImages = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
//  THCudaTensor *grids = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
//  THCudaTensor *gradInputImages = (THCudaTensor *)luaT_checkudata(L, 4, "torch.CudaTensor");
//  THCudaTensor *gradGrids = (THCudaTensor *)luaT_checkudata(L, 5, "torch.CudaTensor");
//  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 6, "torch.CudaTensor");

   //dim3 blocks((gradOutput->size[2]+15)/16, gradOutput->size[1], gradOutput->size[0]);
   dim3 blocks((sz1+15)/16, sz2, sz3);
   dim3 threads(32,16);

   backwardBilinearSampling <false> <<< blocks, threads, 0, /*THCState_getCurrentStream(state)*/stream >>> (
     inputImages_data,
     inputImages_strideBatch,
     inputImages_strideChannels,
     inputImages_strideHeight,
     inputImages_strideWidth,
     gradInputImages_data,
     gradInputImages_strideBatch,
     gradInputImages_strideChannels,
     gradInputImages_strideHeight,
     gradInputImages_strideWidth,
     grids_data,
     grids_strideBatch,
     grids_strideYX,
     grids_strideHeight,
     grids_strideWidth,
     gradGrids_data,
     gradGrids_strideBatch,
     gradGrids_strideYX,
     gradGrids_strideHeight,
     gradGrids_strideWidth,
     masks_data,
     masks_strideBatch,
     masks_strideYX,
     masks_strideHeight,
     masks_strideWidth,
     gradMasks_data,
     gradMasks_strideBatch,
     gradMasks_strideYX,
     gradMasks_strideHeight,
     gradMasks_strideWidth,
     canvas_data,
     canvas_strideBatch,
     canvas_strideYX,
     canvas_strideHeight,
     canvas_strideWidth,
     gradCanvas_data,
     gradCanvas_strideBatch,
     gradCanvas_strideYX,
     gradCanvas_strideHeight,
     gradCanvas_strideWidth,
     gradOutput_data,
     gradOutput_strideBatch,
     gradOutput_strideChannels,
     gradOutput_strideHeight,
     gradOutput_strideWidth,
     inputImages_channels,
     inputImages_height,
     inputImages_width,
     gradOutput_width);



  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in BilinearSampler.updateGradInput: %s\n", hipGetErrorString(err));
    //THError("aborting");
    return 0;
  }
  return 1;
}

int BilinearSamplerBHWD_updateGradInputOnlyGrid_cuda_kernel(
                                        /*gradOutput->size[2]*/int sz1,
                                        /*gradOutput->size[1]*/int sz2,
                                        /*gradOutput->size[0]*/int sz3,
                                        /*THCudaTensor_size(state, inputImages, 3)*/int ic,
                                        /*THCudaTensor_size(state, inputImages, 1)*/int ih,
                                        /*THCudaTensor_size(state, inputImages, 2)*/int iw,
                                        /*THCudaTensor_size(state, gradOutput, 2)*/int gow,
                                        /*THCudaTensor *inputImages*/float *inputImages, int isb, int isc, int ish, int isw,
                                        /*THCudaTensor *grids*/float *grids, int gsb, int gsc, int gsh, int gsw,
                                        /*THCudaTensor *gradGrids*/float *gradGrids, int ggsb, int ggsc, int ggsh, int ggsw,
                                        /*THCudaTensor *gradOutput*/float *gradOutput, int gosb, int gosc, int gosh, int gosw,
                                        /*THCState_getCurrentStream(state)*/hipStream_t stream)
{
//  THCState *state = getCutorchState(L);
//  THCudaTensor *inputImages = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
//  THCudaTensor *grids = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
//  THCudaTensor *gradGrids = (THCudaTensor *)luaT_checkudata(L, 5, "torch.CudaTensor");
//  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 6, "torch.CudaTensor");

   //dim3 blocks((gradOutput->size[2]+15)/16, gradOutput->size[1], gradOutput->size[0]);
   dim3 blocks((sz1+15)/16, sz2, sz3);
   dim3 threads(32,16);

  //  backwardBilinearSampling <true> <<< blocks, threads, 0, /*THCState_getCurrentStream(state)*/stream >>> (
  //                                                     /*THCudaTensor_data(state, inputImages)*/inputImages,
  //                                                     /*THCudaTensor_stride(state, inputImages, 0)*/isb,
  //                                                     /*THCudaTensor_stride(state, inputImages, 3)*/isc,
  //                                                     /*THCudaTensor_stride(state, inputImages, 1)*/ish,
  //                                                     /*THCudaTensor_stride(state, inputImages, 2)*/isw,
  //                                                     0,
  //                                                     0,
  //                                                     0,
  //                                                     0,
  //                                                     0,
  //                                                     /*THCudaTensor_data(state, grids)*/grids,
  //                                                     /*THCudaTensor_stride(state, grids, 0)*/gsb,
  //                                                     /*THCudaTensor_stride(state, grids, 3)*/gsc,
  //                                                     /*THCudaTensor_stride(state, grids, 1)*/gsh,
  //                                                     /*THCudaTensor_stride(state, grids, 2)*/gsw,
  //                                                     /*THCudaTensor_data(state, gradGrids)*/gradGrids,
  //                                                     /*THCudaTensor_stride(state, gradGrids, 0)*/ggsb,
  //                                                     /*THCudaTensor_stride(state, gradGrids, 3)*/ggsc,
  //                                                     /*THCudaTensor_stride(state, gradGrids, 1)*/ggsh,
  //                                                     /*THCudaTensor_stride(state, gradGrids, 2)*/ggsw,
  //                                                     /*THCudaTensor_data(state, gradOutput)*/gradOutput,
  //                                                     /*THCudaTensor_stride(state, gradOutput, 0)*/gosb,
  //                                                     /*THCudaTensor_stride(state, gradOutput, 3)*/gosc,
  //                                                     /*THCudaTensor_stride(state, gradOutput, 1)*/gosh,
  //                                                     /*THCudaTensor_stride(state, gradOutput, 2)*/gosw,
  //                                                     /*THCudaTensor_size(state, inputImages, 3)*/ic,
  //                                                     /*THCudaTensor_size(state, inputImages, 1)*/ih,
  //                                                     /*THCudaTensor_size(state, inputImages, 2)*/iw,
  //                                                     /*THCudaTensor_size(state, gradOutput, 2)*/gow);



  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in BilinearSampler.updateGradInput: %s\n", hipGetErrorString(err));
    //THError("aborting");
    return 0;
  }
  return 1;
}

#ifdef __cplusplus
}
#endif
